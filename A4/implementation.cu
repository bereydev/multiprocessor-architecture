/*
============================================================================
Filename    : implementation.cu
Author      : Jonathan Berezyiat and Lenny Del Zio
SCIPER      : 282962 311240
============================================================================
*/

#include <iostream>
#include <iomanip>
#include <sys/time.h>
#include <hip/hip_runtime.h>
using namespace std;

#define INIT_VALUE 1000
#define INPUT(I, J) input[(I)*length + (J)]
#define OUTPUT(I, J) output[(I)*length + (J)]
#define S_DATA(I, J) sdata[(I)*s_length + (J)]

// CPU Baseline
void array_process(double *input, double *output, int length, int iterations)
{
    double *temp;

    for (int n = 0; n < (int)iterations; n++)
    {
        for (int i = 1; i < length - 1; i++)
        {
            for (int j = 1; j < length - 1; j++)
            {
                 OUTPUT(i, j) = (INPUT(i - 1, j - 1) + INPUT(i - 1, j) + INPUT(i - 1, j + 1) + INPUT(i, j - 1) + INPUT(i, j) + INPUT(i, j + 1) + INPUT(i + 1, j - 1) + INPUT(i + 1, j) + INPUT(i + 1, j + 1)) / 9;
            }
        }
        OUTPUT(length / 2 - 1, length / 2 - 1) = INIT_VALUE;
        OUTPUT(length / 2, length / 2 - 1) = INIT_VALUE;
        OUTPUT(length / 2 - 1, length / 2) = INIT_VALUE;
        OUTPUT(length / 2, length / 2) = INIT_VALUE;

        temp = input;
        input = output;
        output = temp;
    }
}

// GPU functions
// strateforward isolated iteration
__global__ void iterate(double *input, double *output, int length)
{
    int j = (blockIdx.x * blockDim.x) + threadIdx.x;
    int i = (blockIdx.y * blockDim.y) + threadIdx.y;
    if (0 < i && i < length - 1 && 0 < j && j < length - 1)
    {
         OUTPUT(i, j) = (INPUT(i - 1, j - 1) + INPUT(i - 1, j) + INPUT(i - 1, j + 1) + INPUT(i, j - 1) + INPUT(i, j) + INPUT(i, j + 1) + INPUT(i + 1, j - 1) + INPUT(i + 1, j) + INPUT(i + 1, j + 1)) / 9;
    }
    OUTPUT(length / 2 - 1, length / 2 - 1) = INIT_VALUE;
    OUTPUT(length / 2, length / 2 - 1) = INIT_VALUE;
    OUTPUT(length / 2 - 1, length / 2) = INIT_VALUE;
    OUTPUT(length / 2, length / 2) = INIT_VALUE;
}

// Iteration branching on the middle cells to avoid rewriting and avoir performing calculations for the 4 of them
__global__ void iterate_avoid_center(double *input, double *output, int length)
{
    int j = (blockIdx.x * blockDim.x) + threadIdx.x;
    int i = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = (i * length) + j;
    int middle1 = (length / 2 - 1) * length + length / 2 - 1;
    int middle2 = (length / 2) * length + length / 2 - 1;
    int middle3 = (length / 2 - 1) * length + length / 2;
    int middle4 = (length / 2) * length + length / 2;
    if (index == middle1 || index == middle2 || index == middle3 || index == middle4)
    {
        return;
    }
    if (0 < i && i < length - 1 && 0 < j && j < length - 1)
    {
         OUTPUT(i, j) = (INPUT(i - 1, j - 1) + INPUT(i - 1, j) + INPUT(i - 1, j + 1) + INPUT(i, j - 1) + INPUT(i, j) + INPUT(i, j + 1) + INPUT(i + 1, j - 1) + INPUT(i + 1, j) + INPUT(i + 1, j + 1)) / 9;
    }
}

// Iterate using the share memory of the GPU
__global__ void iterate_shared(double *input, double *output, int length)
{
    extern __shared__ double sdata[]; // For the S_DATA macro

    int j = (blockIdx.x * (blockDim.x - 2)) + threadIdx.x;
    int i = (blockIdx.y * (blockDim.y - 2)) + threadIdx.y;
    int s_i = threadIdx.y;
    int s_j = threadIdx.x;
    int s_length = blockDim.x; // For the S_DATA macro
    // Load the shared memory
    if (0 <= i && i <= length - 1 && 0 <= j && j <= length - 1)
    {
        S_DATA(s_i, s_j) = INPUT(i, j);
        __syncthreads();
    }
    if (0 < s_i && s_i < s_length - 1 && 0 < s_j && s_j < s_length - 1)
    {
        if (0 < i && i < length - 1 && 0 < j && j < length - 1)
        {
             OUTPUT(i, j) = (INPUT(i - 1, j - 1) + INPUT(i - 1, j) + INPUT(i - 1, j + 1) + INPUT(i, j - 1) + INPUT(i, j) + INPUT(i, j + 1) + INPUT(i + 1, j - 1) + INPUT(i + 1, j) + INPUT(i + 1, j + 1)) / 9;
        }
    }
    OUTPUT(length / 2 - 1, length / 2 - 1) = INIT_VALUE;
    OUTPUT(length / 2, length / 2 - 1) = INIT_VALUE;
    OUTPUT(length / 2 - 1, length / 2) = INIT_VALUE;
    OUTPUT(length / 2, length / 2) = INIT_VALUE;
}

// GPU Optimized function
void GPU_array_process(double *input, double *output, int length, int iterations)
{
    // Cuda events for calculating elapsed time
    hipEvent_t cpy_H2D_start, cpy_H2D_end, comp_start, comp_end, cpy_D2H_start, cpy_D2H_end;
    hipEventCreate(&cpy_H2D_start);
    hipEventCreate(&cpy_H2D_end);
    hipEventCreate(&cpy_D2H_start);
    hipEventCreate(&cpy_D2H_end);
    hipEventCreate(&comp_start);
    hipEventCreate(&comp_end);

    /* Preprocessing goes here */
    double *gpu_array_in;
    double *gpu_array_out;
    size_t array_size = length * length * sizeof(double);
    // CUDA specific malloc
    hipMalloc((void **)&gpu_array_in, array_size);
    hipMalloc((void **)&gpu_array_out, array_size);

    hipEventRecord(cpy_H2D_start);
    /* Copying array from host to device goes here */
    hipMemcpy((void *)gpu_array_in, (void *)input, array_size, hipMemcpyHostToDevice);
    hipMemcpy((void *)gpu_array_out, (void *)output, array_size, hipMemcpyHostToDevice);

    hipEventRecord(cpy_H2D_end);
    hipEventSynchronize(cpy_H2D_end);

    // Copy array from host to device
    hipEventRecord(comp_start);
    /* GPU calculation goes here */

    // Define a squared thread bloc (chosed option over the commented code under)
    size_t threadBlockSide = 8;
    size_t nbBlockSide = length / threadBlockSide;
    // If not a multiple
    if (length % threadBlockSide != 0)
        nbBlockSide++;

    dim3 thrsPerBlock(threadBlockSide, threadBlockSide);
    dim3 nBlks(nbBlockSide, nbBlockSide);

    // Define the shared memory
    // size_t threadBlockSide_shared = 32;
    // size_t nbBlockSide_shared = length / (threadBlockSide_shared - 2);
    // // If not a multiple
    // if (length % (threadBlockSide_shared - 2) != 0)
    //     nbBlockSide_shared++;

    // size_t smemSize_shared = threadBlockSide_shared * threadBlockSide_shared * sizeof(double);
    // dim3 thrsPerBlock_shared(threadBlockSide_shared, threadBlockSide_shared);
    // dim3 nBlks_shared(nbBlockSide_shared, nbBlockSide_shared);

    // Define a row shaped thread block
    // size_t threadBlockSide_row = length;
    // size_t nbBlockSide_row = 1;
    // if (threadBlockSide_row > 1024)
    // {
    //     threadBlockSide_row = 512;
    //     nbBlockSide_row = length / threadBlockSide_row;
    //     // If not a multiple
    //     if (length % threadBlockSide_row != 0)
    //         nbBlockSide_row++;
    // }
    // dim3 thrsPerBlock_row(threadBlockSide_row, 1);
    // dim3 nBlks_row(nbBlockSide_row, length);

    double *temp;
    for (int n = 0; n < iterations; n++)
    {
        // iterate <<< nBlks, thrsPerBlock >>> (gpu_array_in, gpu_array_out, length);
        // iterate <<< nBlks_row, thrsPerBlock_row >>> (gpu_array_in, gpu_array_out, length);
        // iterate_shared <<< nBlks_shared, thrsPerBlock_shared, smemSize_shared >>> (gpu_array_in, gpu_array_out, length);
        iterate_avoid_center<<<nBlks, thrsPerBlock>>>(gpu_array_in, gpu_array_out, length);

        temp = gpu_array_in;
        gpu_array_in = gpu_array_out;
        gpu_array_out = temp;
    }

    hipEventRecord(comp_end);
    hipEventSynchronize(comp_end);

    hipEventRecord(cpy_D2H_start);
    /* Copying array from device to host goes here */
    hipMemcpy((void *)output, (void *)gpu_array_in, array_size, hipMemcpyDeviceToHost);

    hipEventRecord(cpy_D2H_end);
    hipEventSynchronize(cpy_D2H_end);

    /* Postprocessing goes here */
    hipFree((void *)gpu_array_in);
    hipFree((void *)gpu_array_out);

    float time;
    hipEventElapsedTime(&time, cpy_H2D_start, cpy_H2D_end);
    cout << "Host to Device MemCpy takes " << setprecision(4) << time / INIT_VALUE << "s" << endl;

    hipEventElapsedTime(&time, comp_start, comp_end);
    cout << "Computation takes " << setprecision(4) << time / INIT_VALUE << "s" << endl;

    hipEventElapsedTime(&time, cpy_D2H_start, cpy_D2H_end);
    cout << "Device to Host MemCpy takes " << setprecision(4) << time / INIT_VALUE << "s" << endl;
}